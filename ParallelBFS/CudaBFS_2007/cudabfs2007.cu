#include "hip/hip_runtime.h"
/***
 * @brief This sites https://github.com/siddharths2710/cuda_bfs/blob/master/cuda_bfs/kernel.cu
 * @author XPPGX
 * @date 2023/07/21
*/
#ifndef COMMON
#define COMMON
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#endif

//content
//include C library
extern "C"{
    #include "../../Lib/CSR/CSR.h"
}

//define bBool;
typedef enum{
    no,
    yes
}bBool;

//include cuda library
#include <hip/hip_runtime.h>

__global__ void cudaBfsKernel(int* _csrV, int* _csrE, bBool* _frontier, bBool* _visited, int* _cost, bBool* _done, int _nodeSize){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("tid = %d", tid);
    int neighbor = -1;
    if(tid <= _nodeSize){
        if(_frontier[tid] == yes && _visited[tid] == no){
            printf("%d, ", tid); // tid == nodeID
            _frontier[tid] = no;
            _visited[tid] = yes;
            __syncthreads();

            //visit neighbor
            for(int neighborIndex = _csrV[tid] ; neighborIndex < _csrV[tid + 1] ; neighborIndex ++){
                neighbor = _csrE[neighborIndex];
                if(_visited[neighbor] == no){
                    _cost[neighbor] = _cost[tid] + 1;
                    _frontier[neighbor] = yes;
                    *_done = no;
                }
            }
        }
    }
}

int main(int argc, char* argv[]){
    char* datasetPath = argv[1];
    int startNode = atoi(argv[2]);
//start time 
    struct Graph* adjlist = buildGraph(datasetPath);
    //compute the time of buliding Adjlist
    struct CSR* csr = createCSR(adjlist);
    //compute the time of create csr
    int nodeSize = csr->csrVSize - 1; //actual node size, csr format here will add one space to put the final edge offset.
    showCSR(csr);
    printf("\n\n");
    printf("csr->csrVSize = %d, csr->csrESize = %d\n", csr->csrVSize, csr->csrESize);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("device %d : %s\n", dev, deviceProp.name);
    
    //compute time
#pragma region copyData
    int* cudaCsrV;
    hipMalloc((void**)&cudaCsrV, sizeof(int) * csr->csrVSize);
    hipMemcpy(cudaCsrV, csr->csrV, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);

    int* cudaCsrE;
    hipMalloc((void**)&cudaCsrE, sizeof(int) * csr->csrESize);
    hipMemcpy(cudaCsrE, csr->csrE, sizeof(int) * csr->csrESize, hipMemcpyHostToDevice);

    bBool* hostFrontier = (bBool*)calloc(csr->csrVSize, sizeof(bBool));
    hostFrontier[startNode] = yes;
    bBool* cudaFrontier;
    hipMalloc((void**)&cudaFrontier, sizeof(bBool) * csr->csrVSize);
    hipMemcpy(cudaFrontier, hostFrontier, sizeof(bBool) * csr->csrVSize, hipMemcpyHostToDevice);

    bBool* hostVisited = (bBool*)calloc(csr->csrVSize, sizeof(bBool));
    bBool* cudaVisited;
    hipMalloc((void**)&cudaVisited, sizeof(bBool) * csr->csrVSize);
    hipMemcpy(cudaVisited, hostVisited, sizeof(bBool) * csr->csrVSize, hipMemcpyHostToDevice);

    int* hostCost = (int*)calloc(csr->csrVSize, sizeof(int));
    int* cudaCost;
    hipMalloc((void**)&cudaCost, sizeof(int) * csr->csrVSize);
    hipMemcpy(cudaCost, hostCost, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);
#pragma endregion

#pragma region threadLayout
    dim3 block(32, 1);
    dim3 grid((nodeSize + block.x - 1) / block.x, 1);
    printf("block(x,y) = (%d,%d), grid(x,y) = (%d,%d)\n", block.x, block.y, grid.x, grid.y);
#pragma endregion

    bBool* hostDone = (bBool*)malloc(sizeof(bBool));
    *hostDone = no;
    bBool* cudaDone;
    hipMalloc((void**)&cudaDone, sizeof(bBool));
    hipMemcpy(cudaDone, hostDone, sizeof(bBool), hipMemcpyHostToDevice);
    int count = 0;

    printf("BFS order : ");
    while(*hostDone == no){
        count ++;
        *hostDone = yes;
        hipMemcpy(cudaDone, hostDone, sizeof(bBool), hipMemcpyHostToDevice);
        cudaBfsKernel<<<block, grid>>>(cudaCsrV, cudaCsrE, cudaFrontier, cudaVisited, cudaCost, cudaDone, csr->csrVSize);
        hipMemcpy(hostDone, cudaDone, sizeof(bBool), hipMemcpyDeviceToHost);
    }
    printf("\n");
    hipMemcpy(hostCost, cudaCost, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    printf("Iteration times = %d\n", count);
    printf("Cost : ");
    for(int i = 0 ; i < csr->csrVSize ; i ++){
        printf("cost[%d] = %d\n", i, hostCost[i]);
    }
    //compute whole time
    hipDeviceReset();
}