#include "hip/hip_runtime.h"
/***
 * @author XPPGX
 * @date 2023/07/21
 * @cite The code sites https://github.com/siddharths2710/cuda_bfs/blob/master/cuda_bfs/kernel.cu
*/
#ifndef COMMON
#define COMMON
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#endif

//content

//include C library
extern "C"{
    #ifndef cCSR
    #define cCSR
    #include "../../Lib/CSR/CSR.h"
    #endif

    #ifndef tTime
    #define tTime
    #include "../../Lib/tTime/tTime.h"
    #endif
}

//define
// #define showCost
// #define _DEBUG_
typedef enum{
    no,
    yes
}bBool;

//include cuda library
#include <hip/hip_runtime.h>

__global__ void cudaBfsKernel(int* _csrV, int* _csrE, bBool* _frontier, bBool* _visited, int* _cost, bBool* _done, int _nodeSize, int level){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int neighbor = -1;
    if(tid < _nodeSize){
        if(_frontier[tid] == yes && _visited[tid] == no){
            
            #ifdef _DEBUG_
            printf("%d, ", tid); // tid == nodeID
            #endif

            _frontier[tid] = no;
            _visited[tid] = yes;
            _cost[tid] = level;
            __syncthreads();

            //visit neighbor
            for(int neighborIndex = _csrV[tid] ; neighborIndex < _csrV[tid + 1] ; neighborIndex ++){
                neighbor = _csrE[neighborIndex];
                if(_visited[neighbor] == no){
                    _frontier[neighbor] = yes;
                    *_done = no;
                }
            }
        }
    }
}

int main(int argc, char* argv[]){
    char* datasetPath = argv[1];
    int startNode = atoi(argv[2]);

    double time1 = 0;
    double time2 = 0;
    double startTime = 0;
    
    double timeBuildAdjlist = 0;
    double timeBuildCSR = 0;
    double timeCopyData = 0;
    double timeParallelBFS = 0;
    double timeTotal = 0;

    //Time counter start
    startTime = seconds();
    struct Graph* adjlist = buildGraph(datasetPath);
    time1 = seconds();
    timeBuildAdjlist = time1 - startTime;
    //compute the time of buliding Adjlist
    time1 = seconds();
    struct CSR* csr = createCSR(adjlist);
    time2 = seconds();
    timeBuildCSR = time2 - time1;
    //compute the time of create csr
    int nodeSize = csr->csrVSize - 1; //actual node size, csr format here will add one space to put the final edge offset.
    // showCSR(csr);
    printf("\n\n");
    // printf("csr->csrVSize = %d, csr->csrESize = %d\n", csr->csrVSize, csr->csrESize);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    #ifdef _DEBUG_
    printf("device %d : %s\n", dev, deviceProp.name);
    #endif
    
#pragma region threadLayout
    dim3 block(32, 1);
    dim3 grid((nodeSize + block.x - 1) / block.x, 1);
    printf("block(x,y) = (%d,%d), grid(x,y) = (%d,%d)\n", block.x, block.y, grid.x, grid.y);
#pragma endregion

    //compute time
#pragma region copyData
    time1 = seconds();
    //可以寫個kernel看資料到底有沒有複製到GPU裡面
    int* cudaCsrV;
    hipMalloc((void**)&cudaCsrV, sizeof(int) * csr->csrVSize);
    hipMemcpy(cudaCsrV, csr->csrV, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);


    int* cudaCsrE;
    hipMalloc((void**)&cudaCsrE, sizeof(int) * csr->csrESize);
    hipMemcpy(cudaCsrE, csr->csrE, sizeof(int) * csr->csrESize, hipMemcpyHostToDevice);
    

    bBool* hostFrontier = (bBool*)calloc(csr->csrVSize, sizeof(bBool));
    hostFrontier[startNode] = yes;
    bBool* cudaFrontier;
    hipMalloc((void**)&cudaFrontier, sizeof(bBool) * csr->csrVSize);
    hipMemcpy(cudaFrontier, hostFrontier, sizeof(bBool) * csr->csrVSize, hipMemcpyHostToDevice);

    bBool* hostVisited = (bBool*)calloc(csr->csrVSize, sizeof(bBool));
    bBool* cudaVisited;
    hipMalloc((void**)&cudaVisited, sizeof(bBool) * csr->csrVSize);
    hipMemcpy(cudaVisited, hostVisited, sizeof(bBool) * csr->csrVSize, hipMemcpyHostToDevice);
    

    int* hostCost = (int*)calloc(csr->csrVSize, sizeof(int));
    int* cudaCost;
    hipMalloc((void**)&cudaCost, sizeof(int) * csr->csrVSize);
    hipMemcpy(cudaCost, hostCost, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);
    
    bBool* hostDone = (bBool*)malloc(sizeof(bBool));
    *hostDone = no;
    bBool* cudaDone;
    hipMalloc((void**)&cudaDone, sizeof(bBool));
    hipMemcpy(cudaDone, hostDone, sizeof(bBool), hipMemcpyHostToDevice);
    int count = 0;

    time2 = seconds();

    timeCopyData = time2 - time1;
#pragma endregion

#pragma region algo
    time1 = seconds();

    #ifdef _DEBUG_
    printf("BFS order : ");
    #endif

    while(*hostDone == no){
        *hostDone = yes;

        #ifdef _DEBUG_
        printf("\ncount = %d, { ", count);
        #endif

        hipMemcpy(cudaDone, hostDone, sizeof(bBool), hipMemcpyHostToDevice);
        cudaBfsKernel<<<grid, block>>>(cudaCsrV, cudaCsrE, cudaFrontier, cudaVisited, cudaCost, cudaDone, csr->csrVSize, count);
        hipMemcpy(hostDone, cudaDone, sizeof(bBool), hipMemcpyDeviceToHost);
        
        #ifdef _DEBUG_
        printf("}");
        #endif

        count ++;
    }
    printf("\n");

    time2 = seconds();
    timeParallelBFS = time2 - time1;
#pragma endregion

#pragma region copyData2
    time1 = seconds();
    hipMemcpy(hostCost, cudaCost, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    time2 = seconds();
    timeCopyData = timeCopyData + (time2 - time1);
#pragma endregion

    printf("Iteration times = %d\n", count);

    #ifdef showCost
    printf("Cost :\n");
    int nodeId = 0;
    if(csr->startAtZero == 0){nodeId = 1;}
    for(; nodeId < csr->csrVSize - 1 ; nodeId ++){
        printf("cost[%d] = %d\n", nodeId, hostCost[nodeId]);
    }
    #endif
    //compute whole time
    hipDeviceReset();
    time1 = seconds();
    timeTotal = time1 - startTime;
    
    printf("============================================================\n");
    printf("[Execution Time] Build Adjlist    = %9f\n", timeBuildAdjlist);
    printf("[Execution Time] Build CSR        = %9f\n", timeBuildCSR);
    printf("[Execution Time] Copy Data        = %9f\n", timeCopyData);
    printf("[Execution Time] ParallelBFS algo = %9f\n", timeParallelBFS);
    printf("[Execution Time] TotalTime        = %9f\n", timeTotal);
    printf("============================================================\n");
}